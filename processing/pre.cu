#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "cam.hpp"

#define RECTIFICATION_THREADS 128
#define RECTIFICATION_THREAD_PIXELS 16
#define RECTIFICATION_BLOCKS (((RECTIFIED_IMAGE_HEIGHT + RECTIFICATION_THREAD_PIXELS - 1) / (RECTIFICATION_THREAD_PIXELS) * RECTIFIED_IMAGE_WIDTH + RECTIFICATION_THREADS - 1) / RECTIFICATION_THREADS)

#define TRANSPOSING_THREADS 128
#define TRANSPOSING_THREAD_PIXELS 16
#define TRANSPOSING_BLOCKS (((RECTIFIED_IMAGE_WIDTH + TRANSPOSING_THREAD_PIXELS - 1) / (TRANSPOSING_THREAD_PIXELS) * RECTIFIED_IMAGE_HEIGHT + TRANSPOSING_THREADS - 1) / TRANSPOSING_THREADS)

#define VERTICAL_INTEGRATION_THREADS 128
#define VERTICAL_INTEGRATION_BLOCKS ((RECTIFIED_IMAGE_WIDTH + 1 + VERTICAL_INTEGRATION_THREADS - 1) / VERTICAL_INTEGRATION_THREADS)
#define HORIZONTAL_INTEGRATION_THREADS 416
#define HORIZONTAL_INTEGRATION_BLOCKS RECTIFIED_IMAGE_HEIGHT

unsigned char *d_di;
size_t d_dip;
texture<unsigned char, hipTextureType2D, hipReadModeNormalizedFloat> t_di;
size_t t_dio;

unsigned char *d_lri;
unsigned char *d_rri;

int *d_lrii;
size_t d_lriip;
hipTextureObject_t t_lriii;

int *d_rrii;
size_t d_rriip;
hipTextureObject_t t_rriii;

unsigned char *d_ltri;
unsigned char *d_rtri;

template<int w, int h>
__global__ void rectifyLeftImage(unsigned char *d_ri, unsigned char *d_di, int t_dio);
template<int w, int h>
__global__ void rectifyRightImage(unsigned char *d_ri, unsigned char *d_di, int t_dio);
template<int w, int h>
__global__ void integrateImageVertically(int *d_rii, int d_riip, unsigned char *d_ri);
template<int w>
__global__ void integrateImageHorizontally(int *d_rii, int d_riip);
template<int w, int h>
__global__ void transposeImage(unsigned char *d_tri, unsigned char *d_ri);

#include <iostream>

void initializePreprocessing() {
	hipMallocPitch(&d_di, &d_dip, DISTORTED_IMAGE_WIDTH * sizeof(unsigned char), DISTORTED_IMAGE_HEIGHT);
	t_di.addressMode[0] = hipAddressModeBorder;
	t_di.addressMode[1] = hipAddressModeBorder;
	t_di.filterMode = hipFilterModeLinear;
	hipChannelFormatDesc textureChannelDescription = hipCreateChannelDesc<unsigned char>();
	hipBindTexture2D(&t_dio, &t_di, d_di, &textureChannelDescription, DISTORTED_IMAGE_WIDTH, DISTORTED_IMAGE_HEIGHT, d_dip);
	d_dip /= sizeof(unsigned char);
	t_dio /= sizeof(unsigned char);
	hipMalloc(&d_lri, RECTIFIED_IMAGE_WIDTH * RECTIFIED_IMAGE_HEIGHT * sizeof(unsigned char));
	hipMalloc(&d_rri, RECTIFIED_IMAGE_WIDTH * RECTIFIED_IMAGE_HEIGHT * sizeof(unsigned char));
	hipMallocPitch(&d_lrii, &d_lriip, (RECTIFIED_IMAGE_WIDTH + 1) * sizeof(int), RECTIFIED_IMAGE_HEIGHT + 1);
	textureChannelDescription = hipCreateChannelDesc<int>();
	hipResourceDesc textureResourceDescription;
	memset(&textureResourceDescription, 0, sizeof(textureResourceDescription));
	textureResourceDescription.resType = hipResourceTypePitch2D;
	textureResourceDescription.res.pitch2D.devPtr = d_lrii;
	textureResourceDescription.res.pitch2D.desc = textureChannelDescription;
	textureResourceDescription.res.pitch2D.width = RECTIFIED_IMAGE_WIDTH + 1;
	textureResourceDescription.res.pitch2D.height = RECTIFIED_IMAGE_HEIGHT + 1;
	textureResourceDescription.res.pitch2D.pitchInBytes = d_lriip;
	d_lriip /= sizeof(int);
	hipTextureDesc textureDescription;
	memset(&textureDescription, 0, sizeof(textureDescription));
	textureDescription.addressMode[0] = hipAddressModeClamp;
	textureDescription.addressMode[1] = hipAddressModeClamp;
	textureDescription.filterMode = hipFilterModePoint;
	textureDescription.readMode = hipReadModeElementType;
	textureDescription.normalizedCoords = false;
	memset(&t_lriii, 0, sizeof(t_lriii));
	hipCreateTextureObject(&t_lriii, &textureResourceDescription, &textureDescription, NULL);
	hipMallocPitch(&d_rrii, &d_rriip, (RECTIFIED_IMAGE_WIDTH + 1) * sizeof(int), RECTIFIED_IMAGE_HEIGHT + 1);
	textureResourceDescription.res.pitch2D.devPtr = d_rrii;
	textureResourceDescription.res.pitch2D.pitchInBytes = d_rriip;
	d_rriip /= sizeof(int);
	memset(&t_rriii, 0, sizeof(t_rriii));
	hipCreateTextureObject(&t_rriii, &textureResourceDescription, &textureDescription, NULL);
	hipMalloc(&d_ltri, RECTIFIED_IMAGE_WIDTH * RECTIFIED_IMAGE_HEIGHT * sizeof(unsigned char));
	hipMalloc(&d_rtri, RECTIFIED_IMAGE_WIDTH * RECTIFIED_IMAGE_HEIGHT * sizeof(unsigned char));
}

void rectifyImages(unsigned char *l, unsigned char *r, unsigned char *o) {
	hipMemcpy2D(d_di, d_dip, l, DISTORTED_IMAGE_WIDTH * sizeof(unsigned char), DISTORTED_IMAGE_WIDTH * sizeof(unsigned char), DISTORTED_IMAGE_HEIGHT, hipMemcpyHostToDevice);
	rectifyLeftImage<RECTIFIED_IMAGE_WIDTH, RECTIFIED_IMAGE_HEIGHT><<<RECTIFICATION_BLOCKS, RECTIFICATION_THREADS>>>(d_lri, d_di, t_dio);
	hipMemcpy2D(d_di, d_dip, r, DISTORTED_IMAGE_WIDTH * sizeof(unsigned char), DISTORTED_IMAGE_WIDTH * sizeof(unsigned char), DISTORTED_IMAGE_HEIGHT, hipMemcpyHostToDevice);
	rectifyRightImage<RECTIFIED_IMAGE_WIDTH, RECTIFIED_IMAGE_HEIGHT><<<RECTIFICATION_BLOCKS, RECTIFICATION_THREADS>>>(d_rri, d_di, t_dio);
}

void integrateImages() {
	integrateImageVertically<RECTIFIED_IMAGE_WIDTH, RECTIFIED_IMAGE_HEIGHT><<<VERTICAL_INTEGRATION_BLOCKS, VERTICAL_INTEGRATION_THREADS>>>(d_lrii, d_lriip, d_lri);
	integrateImageHorizontally<RECTIFIED_IMAGE_WIDTH><<<HORIZONTAL_INTEGRATION_BLOCKS, HORIZONTAL_INTEGRATION_THREADS>>>(d_lrii, d_lriip);  
	integrateImageVertically<RECTIFIED_IMAGE_WIDTH, RECTIFIED_IMAGE_HEIGHT><<<VERTICAL_INTEGRATION_BLOCKS, VERTICAL_INTEGRATION_THREADS>>>(d_rrii, d_rriip, d_rri);
	integrateImageHorizontally<RECTIFIED_IMAGE_WIDTH><<<HORIZONTAL_INTEGRATION_BLOCKS, HORIZONTAL_INTEGRATION_THREADS>>>(d_rrii, d_rriip);	
}

void transposeImages() {
	transposeImage<RECTIFIED_IMAGE_WIDTH, RECTIFIED_IMAGE_HEIGHT><<<TRANSPOSING_BLOCKS, TRANSPOSING_THREADS>>>(d_ltri, d_lri);
	transposeImage<RECTIFIED_IMAGE_WIDTH, RECTIFIED_IMAGE_HEIGHT><<<TRANSPOSING_BLOCKS, TRANSPOSING_THREADS>>>(d_rtri, d_rri);
}

template<int w, int h>
__global__ void rectifyLeftImage(unsigned char *d_ri, unsigned char *d_di, int t_dio) {
	int l_c = threadIdx.x + blockIdx.x * blockDim.x;
	int l_r = l_c / w;
	l_c -= l_r * w;
	l_r *= RECTIFICATION_THREAD_PIXELS;
	for (int l_ro = 0; l_ro < RECTIFICATION_THREAD_PIXELS && l_r + l_ro < h; l_ro++) {
		float l_ud = (l_c + LEFT_RECTIFIED_COLUMN) * LEFT_HOMOGRAPHY_20 + (l_r + l_ro + TOP_RECTIFIED_ROW) * LEFT_HOMOGRAPHY_21 + LEFT_HOMOGRAPHY_22;
		float l_ux = ((l_c + LEFT_RECTIFIED_COLUMN) * LEFT_HOMOGRAPHY_00 + (l_r + l_ro + TOP_RECTIFIED_ROW) * LEFT_HOMOGRAPHY_01 + LEFT_HOMOGRAPHY_02) / l_ud;
		float l_uy = ((l_c + LEFT_RECTIFIED_COLUMN) * LEFT_HOMOGRAPHY_10 + (l_r + l_ro + TOP_RECTIFIED_ROW) * LEFT_HOMOGRAPHY_11 + LEFT_HOMOGRAPHY_12) / l_ud;
		float l_sr = l_ux * l_ux + l_uy * l_uy;
		float l_df = l_sr * l_sr * SECOND_LEFT_DISTORTION_COEFFICIENT + l_sr * FIRST_LEFT_DISTORTION_COEFFICIENT + 1.0;
		float l_dc = l_ux * l_df * LEFT_HORIZONTAL_FOCAL_LENGTH + LEFT_HORIZONTAL_PRINCIPAL_POINT;
		float l_dr = l_uy * l_df * LEFT_VERTICAL_FOCAL_LENGTH + LEFT_VERTICAL_PRINCIPAL_POINT;
		//offset texture coordinates by 0.5?
		d_ri[l_c + (l_r + l_ro) * w] = (unsigned char) (tex2D(t_di, l_dc + t_dio, l_dr) * UCHAR_MAX);
	}
}

template<int w, int h>
__global__ void rectifyRightImage(unsigned char *d_ri, unsigned char *d_di, int t_dio) {
	int l_c = threadIdx.x + blockIdx.x * blockDim.x;
	int l_r = l_c / w;
	l_c -= l_r * w;
	l_r *= RECTIFICATION_THREAD_PIXELS;
	for (int l_ro = 0; l_ro < RECTIFICATION_THREAD_PIXELS && l_r + l_ro < h; l_ro++) {
		float l_ud = (l_c + LEFT_RECTIFIED_COLUMN) * RIGHT_HOMOGRAPHY_20 + (l_r + l_ro + TOP_RECTIFIED_ROW) * RIGHT_HOMOGRAPHY_21 + RIGHT_HOMOGRAPHY_22;
		float l_ux = ((l_c + LEFT_RECTIFIED_COLUMN) * RIGHT_HOMOGRAPHY_00 + (l_r + l_ro + TOP_RECTIFIED_ROW) * RIGHT_HOMOGRAPHY_01 + RIGHT_HOMOGRAPHY_02) / l_ud;
		float l_uy = ((l_c + LEFT_RECTIFIED_COLUMN) * RIGHT_HOMOGRAPHY_10 + (l_r + l_ro + TOP_RECTIFIED_ROW) * RIGHT_HOMOGRAPHY_11 + RIGHT_HOMOGRAPHY_12) / l_ud;
		float l_sr = l_ux * l_ux + l_uy * l_uy;
		float l_df = l_sr * l_sr * SECOND_RIGHT_DISTORTION_COEFFICIENT + l_sr * FIRST_RIGHT_DISTORTION_COEFFICIENT + 1.0;
		float l_dc = l_ux * l_df * RIGHT_HORIZONTAL_FOCAL_LENGTH + RIGHT_HORIZONTAL_PRINCIPAL_POINT;
		float l_dr = l_uy * l_df * RIGHT_VERTICAL_FOCAL_LENGTH + RIGHT_VERTICAL_PRINCIPAL_POINT;
		d_ri[l_c + (l_r + l_ro) * w] = (unsigned char) (tex2D(t_di, l_dc + t_dio, l_dr) * UCHAR_MAX);
	}
}

template<int w, int h>
__global__ void integrateImageVertically(int *d_rii, int d_riip, unsigned char *d_ri) {
	int l_c = threadIdx.x + blockIdx.x * blockDim.x;
	if (l_c < w + 1) {
		d_rii[l_c] = 0;
		int l_s = 0;
		if (l_c > 0)
			for (int l_r = 1; l_r < h + 1; l_r++) {
				l_s += d_ri[l_c - 1 + (l_r - 1) * w];
				d_rii[l_c + l_r * d_riip] = l_s;
			}
		else
			for (int l_r = 1; l_r < h + 1; l_r++)
				d_rii[l_r * d_riip] = 0;
	}
}

template<int w>
__global__ void integrateImageHorizontally(int *d_rii, int d_riip) {
	int l_r = blockIdx.x + 1;
	volatile __shared__ int s_b[2][w + 1];
	for (int l_c = threadIdx.x; l_c < w + 1; l_c += blockDim.x)
		s_b[0][l_c] = d_rii[l_c + l_r * d_riip];	
	int l_i = 0;
	for (int l_co = 1; w - l_co > -1; l_co <<= 1) {
		__syncthreads();
		for (int l_c = threadIdx.x; l_c < w + 1; l_c += blockDim.x)
			if (l_c - l_co > -1)
				s_b[1 - l_i][l_c] = s_b[l_i][l_c] + s_b[l_i][l_c - l_co];
			else
				s_b[1 - l_i][l_c] = s_b[l_i][l_c];
		l_i = 1 - l_i;
	}
	for (int l_c = threadIdx.x; l_c < w + 1; l_c += blockDim.x)
		d_rii[l_c + l_r * d_riip] = s_b[l_i][l_c];
}

template<int w, int h>
__global__ void transposeImage(unsigned char *d_tri, unsigned char *d_ri) {
	int l_r = threadIdx.x + blockIdx.x * blockDim.x;
	int l_c = l_r / h;
	l_r -= l_c * h;
	l_c *= TRANSPOSING_THREAD_PIXELS;
	#pragma unroll
	for (int l_ro = 0; l_ro < TRANSPOSING_THREAD_PIXELS && l_c + l_ro < w; l_ro++)
		d_tri[l_r + (l_c + l_ro) * h] = d_ri[l_c + l_ro + l_r * w];
}